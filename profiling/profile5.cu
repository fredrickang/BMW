
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <signal.h>
#include <time.h>

#include <fcntl.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <string.h>
#include <sched.h>
#include <float.h>
#include <signal.h>
#include <sys/time.h>
#include <sys/resource.h>


// Allocate 10GB (unit of allocation is not matter)






int request_fd = -1;
int decision_fd = -1;
int register_fd = -1;
int Sync = 1;
struct timespec *release_time = NULL;

typedef struct _MSG_PACKET_REG{
    int regist;
    int pid;
    double period;
}reg_msg;

int communicate(int ack){
    int decision = 0; 
    
    if( write(request_fd, &ack, sizeof(int)*1) == -1){
        perror("Request Send :");
        exit(-1);
    }
    if(Sync){
        release_time = (struct timespec *)malloc(sizeof(struct timespec));
        if(read(decision_fd, release_time, sizeof(struct timespec))< 0){
            perror("release time");
        }
        Sync = 0;
    }

    if(read(decision_fd, &decision, sizeof(int)*1) == -1){
        perror("Decision Recv :");
        exit(-1);
    }
    return decision;
}

int main(void){
    
    if( (register_fd = open("/tmp/scheduler", O_WRONLY)) < 0){
        perror("Opening Registration channel");
        exit(-1);
    }

    reg_msg * reg = (reg_msg *)malloc(sizeof(reg_msg));
    reg->regist = 1;
    reg->pid = getpid();
    reg->period = 104;

    if(write(register_fd, reg, sizeof(reg_msg)) < 0){
        perror("Registrating: ");
        exit(-1);
    }

    fprintf(stderr, "Registrated (%d)\n", getpid());

    char request[50];
    char decision[50];

    snprintf(request, 50, "/tmp/sch_request_%d",getpid());
    snprintf(decision, 50, "/tmp/sch_decision_%d",getpid());

    while( (request_fd = open(request, O_WRONLY)) < 0);
    while( (decision_fd = open(decision, O_RDONLY)) < 0);
    fprintf(stderr, "==%d== comms open!\n",getpid());

    void *tmp;
    hipMalloc(&tmp, 1);
    int ack = 99;
    if( write(request_fd, &ack, sizeof(int)*1) == -1){
        perror("Request Send :");
        exit(-1);
    }
    if(read(decision_fd, &ack, sizeof(int)*1) == -1){
        perror("Decision Recv :");
        exit(-1);
    }

    communicate(0);
    int ret;
    // Do Things 
    int *** chunks = (int ***)malloc(sizeof(int **)*1);
    for(int i = 0; i < 1; i++){
        chunks[i] = (int **)malloc(sizeof(int *)*3);
        for(int j = 0; j < 3; j++){
            ret = hipMalloc(&chunks[i][j], 4294967294);
            if(ret != 0) printf("Error: %d\n", ret);
        }
    }
    if (write(request_fd, &ack, sizeof(int)) == -1){
        perror("Request Send:");
        exit(-1);
    }
    communicate(0);
}
