#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


int main(void){
    void *d_a[10];
    for(int i = 0; i < 10; i++){
        hipMalloc(&d_a[i], sizeof(char)*1000);
        printf("%d th address: %p\n",i, d_a[i]);
    }
    printf("%d\n",(int *)d_a[1] -(int *)d_a[0]);

    hipFree(d_a[1]);
    int *c;
    hipMalloc(&c, sizeof(int)*1024);
    printf("target address: %p\n", c);

    return 0;
}