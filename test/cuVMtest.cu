#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


int main(void){
    int ret;
    int *a, *d_a, *d_b, *b;
    a = (int *)malloc(sizeof(int)*10);
    b = (int *)malloc(sizeof(int)*10);

    hipMalloc(&d_a, sizeof(int)*10);
    
    for(int i = 0; i < 10; i++) a[i] = i;

    hipMemcpy(d_a, a, sizeof(int)*10, hipMemcpyHostToDevice);
    d_b = d_a;

    hipFree(d_a);
    hipMalloc(&d_b, sizeof(int)*10);
    hipMemcpy(d_b, a, sizeof(int)*10, hipMemcpyHostToDevice);

    ret = hipMemcpy(b, d_a, sizeof(int)*10, hipMemcpyDeviceToHost);
    printf("cudamemcpy ret: %d\n", ret);
    for(int i =0; i < 10; i++){
        printf("%d,",b[i]);
    }
    printf("\n");

    return 0;
}